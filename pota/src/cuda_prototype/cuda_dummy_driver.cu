#include "hip/hip_runtime.h"
#define TINYEXR_IMPLEMENTATION
#include "../tinyexr.h"

#include "../../../Eigen/Eigen/Core"
#include "../../../Eigen/Eigen/LU"
#include <fstream>
#include <iostream>
#include <vector>



#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}



std::string replace_first_occurence(std::string& s, const std::string& toReplace, const std::string& replaceWith) {
    std::size_t pos = s.find(toReplace);
    if (pos == std::string::npos) return s;
    return s.replace(pos, toReplace.length(), replaceWith);
}

void save_to_exr_rgba(std::vector<float> img, std::string filename, unsigned xres, unsigned yres) {
  EXRHeader header;
  InitEXRHeader(&header);

  EXRImage image;
  InitEXRImage(&image);
  image.num_channels = 4;
  image.width = xres;
  image.height = yres;

  std::vector<float> images[4];
  images[0].resize(xres * yres);
  images[1].resize(xres * yres);
  images[2].resize(xres * yres);
  images[3].resize(xres * yres);

  for (unsigned int i = 0; i < xres * yres; i++) {
    images[0][i] = img[4*i+0];
    images[1][i] = img[4*i+1];
    images[2][i] = img[4*i+2];
    images[3][i] = img[4*i+3];
  }

  float* image_ptr[4];
  image_ptr[0] = &(images[3].at(0)); // A
  image_ptr[1] = &(images[2].at(0)); // B
  image_ptr[2] = &(images[1].at(0)); // G
  image_ptr[3] = &(images[0].at(0)); // R

  image.images = (unsigned char**)image_ptr;
  header.num_channels = 4;
  header.channels = (EXRChannelInfo *)malloc(sizeof(EXRChannelInfo) * header.num_channels);
  strncpy(header.channels[0].name, "A", 255); header.channels[0].name[strlen("A")] = '\0';
  strncpy(header.channels[1].name, "B", 255); header.channels[1].name[strlen("B")] = '\0';
  strncpy(header.channels[2].name, "G", 255); header.channels[2].name[strlen("G")] = '\0';
  strncpy(header.channels[3].name, "R", 255); header.channels[3].name[strlen("R")] = '\0';

  header.pixel_types = (int *)malloc(sizeof(int) * header.num_channels); 
  header.requested_pixel_types = (int *)malloc(sizeof(int) * header.num_channels);
  for (int i = 0; i < header.num_channels; i++) {
    header.pixel_types[i] = TINYEXR_PIXELTYPE_FLOAT; // pixel type of input image
    header.requested_pixel_types[i] = TINYEXR_PIXELTYPE_HALF; // pixel type of output image to be stored in .EXR
  }
  
  header.compression_type = TINYEXR_COMPRESSIONTYPE_ZIP;

  const char* err;
  int ret = SaveEXRImageToFile(&image, &header, filename.c_str(), &err);
  if (ret != TINYEXR_SUCCESS) {
    std::cout << "[LENTIL BIDIRECTIONAL TL] Error when saving exr: " << err << std::endl;
  }
}


// Improved concentric mapping code by Dave Cline [peter shirley´s blog]
// maps points on the unit square onto the unit disk uniformly
__device__ void concentricDiskSample(float ox, float oy, Eigen::Vector2d &lens) {
    if (ox == 0.0 && oy == 0.0){
        lens(0) = 0.0;
        lens(1) = 0.0;
        return;
    }

    float phi, r;

    // switch coordinate space from [0, 1] to [-1, 1]
    const float a = 2.0 * ox - 1.0;
    const float b = 2.0 * oy - 1.0;

    if ((a * a) > (b * b)){
        r = a;
        phi = 0.78539816339 * (b / a);
    }
    else {
        r = b;
        phi = (1.57079632679) - ((0.78539816339) * (a / b));
    }


    bool fast_trigo = false;

    const float cos_phi = cosf(phi);
    const float sin_phi =  sinf(phi);
    lens(0) = r * cos_phi;
    lens(1) = r * sin_phi;
}


__device__ float thinlens_get_image_dist_focusdist(const float focal_length, const float focus_distance){
    return (-focal_length * -focus_distance) / (-focal_length + -focus_distance);
}



// https://github.com/nvpro-samples/optix_advanced_samples/blob/master/src/optixIntroduction/optixIntro_06/shaders/random_number_generators.h
// Tiny Encryption Algorithm (TEA) to calculate a the seed per launch index and iteration.
template<unsigned int N>
__device__ unsigned int tea(const unsigned int val0, const unsigned int val1)
{
  unsigned int v0 = val0;
  unsigned int v1 = val1;
  unsigned int s0 = 0;

  for (unsigned int n = 0; n < N; ++n)
  {
    s0 += 0x9e3779b9;
    v0 += ((v1 << 4) + 0xA341316C) ^ (v1 + s0) ^ ((v1 >> 5) + 0xC8013EA4);
    v1 += ((v0 << 4) + 0xAD90777D) ^ (v0 + s0) ^ ((v0 >> 5) + 0x7E95761E);
  }
  return v0;
}


// https://github.com/nvpro-samples/optix_advanced_samples/blob/master/src/optixIntroduction/optixIntro_06/shaders/random_number_generators.h
// Return a random sample in the range [0, 1) with a simple Linear Congruential Generator.
__device__ float rng(unsigned int& previous)
{
  previous = previous * 1664525u + 1013904223u;
  
  return float(previous & 0X00FFFFFF) / float(0x01000000u); // Use the lower 24 bits.
  // return float(previous >> 8) / float(0x01000000u);      // Use the upper 24 bits
}



inline int ceil_to_power_2(int x) {
    if (x < 2) return 1;
    return (int) std::pow(2, (int) std::log2(x-1) + 1);
}




inline float clamp_min(float in, const float min) {
    if (in < min) in = min;
    return in;
}

inline float clamp(float in, const float min, const float max) {
    if (in < min) in = min;
    if (in > max) in = max;
    return in;
}


__global__ void trace_backwards(Eigen::Vector4d *image, Eigen::Vector4d *image_unredist, Eigen::Vector4d *image_redist,
                                      float *redist_weight_per_pixel, float *unredist_weight_per_pixel, float *zbuffer,
                                      const Eigen::Vector3d &sample_pos_cs, const float &focal_length, const float &aperture_radius, 
                                      const float &focus_distance, const float &sensor_width, const float &frame_aspect_ratio,
                                      const int &xres, const int &yres,
                                      const Eigen::Vector4d &sample, const float &inv_density,
                                      int &iter) {

  int trycount = 0;
  bool success = false;
  while(!success && ++trycount < 10){
    
    const Eigen::Vector3d camera_space_sample_position_mb = sample_pos_cs;
    const float image_dist_samplepos_mb = (-focal_length * camera_space_sample_position_mb(2)) / (-focal_length + camera_space_sample_position_mb(2));



    // either get uniformly distributed points on the unit disk or bokeh image
    Eigen::Vector2d unit_disk(0, 0);
    unsigned int seed = tea<8>(threadIdx.x + blockIdx.x * blockDim.x, iter + trycount);
    concentricDiskSample(rng(seed), rng(seed), unit_disk);
    
    // ray through center of lens
    Eigen::Vector3d dir_tobase = camera_space_sample_position_mb.normalized();
    float samplepos_image_intersection = std::abs(image_dist_samplepos_mb/dir_tobase(2));
    Eigen::Vector3d samplepos_image_point = dir_tobase * samplepos_image_intersection;

    // depth of field
    Eigen::Vector3d lens(unit_disk(0) * aperture_radius, unit_disk(1) * aperture_radius, 0.0);
    Eigen::Vector3d dir_from_lens_to_image_sample = samplepos_image_point - lens;
    dir_from_lens_to_image_sample.normalize();
    float focusdist_intersection = std::abs(thinlens_get_image_dist_focusdist(focal_length, focus_distance)/dir_from_lens_to_image_sample(2));
    

    Eigen::Vector3d focusdist_image_point = lens + dir_from_lens_to_image_sample*focusdist_intersection;
    
    // takes care of correct screenspace coordinate mapping
    Eigen::Vector2d sensor_position(focusdist_image_point(0) / focusdist_image_point(2),
                                    focusdist_image_point(1) / focusdist_image_point(2));
    sensor_position /= (sensor_width*0.5)/-focal_length;


    // optical vignetting
    Eigen::Vector3d dir_lens_to_P = camera_space_sample_position_mb - lens;
    dir_lens_to_P.normalize();


    // convert sensor position to pixel position
    float frame_aspect_ratio_tmp = frame_aspect_ratio;
    const float pixel_x = (( sensor_position(0) + 1.0) / 2.0) * xres;
    const float pixel_y = ((-sensor_position(1) * frame_aspect_ratio_tmp + 1.0) / 2.0) * yres;

    // if outside of image
    if ((pixel_x >= xres) || (pixel_x < 0) || (pixel_y >= yres) || (pixel_y < 0)) continue;

    // write sample to image
    unsigned pixelnumber = static_cast<int>(xres * floor(pixel_y) + floor(pixel_x));

    
    Eigen::Vector4d rgba_energy = sample / (double)(blockDim.x * gridDim.x);
    image_redist[pixelnumber] += rgba_energy * inv_density;
    redist_weight_per_pixel[pixelnumber] += inv_density / double(blockDim.x * gridDim.x);
    
    success = true; 
  }  
  return;
}







void allthestuff() {
  std::ifstream infile("/home/cactus/lentil/pota/tests/cuda/sampledata.txt");
  float sample_r, sample_g, sample_b, sample_a, depth, sample_pos_ws_x, sample_pos_ws_y, sample_pos_ws_z;
  std::vector<Eigen::Vector4d> sample_list;
  std::vector<Eigen::Vector3d> pos_ws_list;
  std::vector<float> depth_list;
  while (infile >> sample_r >> sample_g >> sample_b >> sample_a >> depth >> sample_pos_ws_x >> sample_pos_ws_y >> sample_pos_ws_z)
  {
      sample_list.push_back(Eigen::Vector4d(sample_r, sample_g, sample_b, sample_a));
      depth_list.push_back(depth);
      pos_ws_list.push_back(Eigen::Vector3d(sample_pos_ws_x, sample_pos_ws_y, sample_pos_ws_z));
  }




  int xres = 1920;
  int yres = 1080;
  int framenumber = 1;
  int aa_samples = 4;
  

  Eigen::Vector4d *image, *image_unredist, *image_redist;
  float *redist_weight_per_pixel, *unredist_weight_per_pixel, *zbuffer;
  checkCudaErrors(hipMallocManaged(&image, xres*yres*sizeof(Eigen::Vector4d)));
  checkCudaErrors(hipMallocManaged(&image_unredist, xres*yres*sizeof(Eigen::Vector4d)));
  checkCudaErrors(hipMallocManaged(&image_redist, xres*yres*sizeof(Eigen::Vector4d)));
  checkCudaErrors(hipMallocManaged(&redist_weight_per_pixel, xres*yres*sizeof(float)));
  checkCudaErrors(hipMallocManaged(&unredist_weight_per_pixel, xres*yres*sizeof(float)));
  checkCudaErrors(hipMallocManaged(&zbuffer, xres*yres*sizeof(float)));




  float sensor_width;
  float focal_length;
  float fstop;
  float focus_distance;
  float aperture_radius;
  float abb_spherical;
  float circle_to_square;
  float bokeh_anamorphic;
  std::string bidir_output_path;
  unsigned int bidir_sample_mult;

  sensor_width = 36.0;
  focal_length = 50.0;
  focal_length = clamp_min(focal_length, 0.01);
  fstop = 1.4;
  fstop = clamp_min(fstop, 0.01);
  focus_distance = 35.0;
  aperture_radius = (focal_length / (2.0 * fstop)) / 10.0;
  bidir_output_path = "/home/cactus/lentil/pota/tests/cuda/cuda.tl.<aov>.<frame>.exr";
  abb_spherical = 0.5;
  abb_spherical = clamp(abb_spherical, 0.001, 0.999);
  circle_to_square = 0.0;
  circle_to_square = clamp(circle_to_square, 0.01, 0.99);
  bokeh_anamorphic = 1.0;
  bokeh_anamorphic = clamp(bokeh_anamorphic, 0.01, 99999.0);
  bidir_sample_mult = 10;


  




  const float d_xres = (float)xres;
  const float d_yres = (float)yres;
  const float frame_aspect_ratio = d_xres/d_yres;




  int *xres_device, *yres_device;
  float *focal_length_device, *aperture_radius_device, *focus_distance_device, *sensor_width_device, *frame_aspect_ratio_device;
  checkCudaErrors(hipMalloc((void **)&xres_device, sizeof(int)));
  checkCudaErrors(hipMalloc((void **)&yres_device, sizeof(int)));
  checkCudaErrors(hipMalloc((void **)&focal_length_device, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&aperture_radius_device, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&focus_distance_device, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&sensor_width_device, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&frame_aspect_ratio_device, sizeof(float)));
  checkCudaErrors(hipMemcpy(xres_device, &xres, sizeof(int), hipMemcpyHostToDevice));	
  checkCudaErrors(hipMemcpy(yres_device, &yres, sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(focal_length_device, &focal_length, sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(aperture_radius_device, &aperture_radius, sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(focus_distance_device, &focus_distance, sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(sensor_width_device, &sensor_width, sizeof(float), hipMemcpyHostToDevice));	
  checkCudaErrors(hipMemcpy(frame_aspect_ratio_device, &frame_aspect_ratio, sizeof(float), hipMemcpyHostToDevice));




  for (int i=0; i<sample_list.size(); ++i) {

    Eigen::Vector4d sample = sample_list[i];
    
    const Eigen::Vector3d sample_pos_cs = pos_ws_list[i];
    float depth = depth_list[i];
    const float inv_density = 1.0/16.0;
    
    const Eigen::Vector4d sample_transmission = Eigen::Vector4d(0,0,0,0);
    bool transmitted_energy_in_sample = ((sample_transmission(0)) > 0.0);
    if (transmitted_energy_in_sample){
      sample(0) -= sample_transmission(0);
      sample(1) -= sample_transmission(1);
      sample(2) -= sample_transmission(2);
    }




      // Eigen::Matrix4d world_to_camera_matrix;
      // world_to_camera_matrix << 1.0, 0.0, 0.0, 0.0,
      //                           0.0, 1.0, 0.0, 0.0,
      //                           0.0, 0.0, 1.0, -75.0,
      //                           0.0, 0.0, 0.0, 1.0;

      const Eigen::Vector3d camera_space_sample_position_static = sample_pos_cs;
      
      const float image_dist_samplepos = (-focal_length * camera_space_sample_position_static(2)) / (-focal_length + camera_space_sample_position_static(2));
      const float image_dist_focusdist = (-focal_length * -focus_distance) / (-focal_length + -focus_distance);
      float circle_of_confusion = std::abs((aperture_radius * (image_dist_samplepos - image_dist_focusdist))/image_dist_samplepos); // coc diameter
      


      const float coc_squared_pixels = std::pow(circle_of_confusion * yres, 2) * bidir_sample_mult * 0.003; // pixel area as baseline for sample count
      // if (std::pow(circle_of_confusion * yres, 2) < std::pow(15, 2)) goto no_redist; // 15^2 px minimum coc
      int samples = std::ceil(coc_squared_pixels / (double)std::pow(aa_samples, 2)); // aa_sample independence
      samples = clamp(samples, 100, 1000000);
      int blocks = ceil_to_power_2(samples); // number of blocks
      // std::cout << samples << std::endl;
      int cudathreads = 64;



    // allocate & copy device variables
    float *inv_density_device;
    Eigen::Vector4d *sample_device;
    Eigen::Vector3d *sample_pos_cs_device;
    int *iter_device;
    checkCudaErrors(hipMalloc((void **)&inv_density_device, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&sample_device, sizeof(Eigen::Vector4d)));
    checkCudaErrors(hipMalloc((void **)&sample_pos_cs_device, sizeof(Eigen::Vector3d)));
    checkCudaErrors(hipMalloc((void **)&iter_device, sizeof(int)));
    checkCudaErrors(hipMemcpy(inv_density_device, &inv_density, sizeof(float), hipMemcpyHostToDevice));	
    checkCudaErrors(hipMemcpy(sample_device, &sample, sizeof(Eigen::Vector4d), hipMemcpyHostToDevice));	
    checkCudaErrors(hipMemcpy(sample_pos_cs_device, &sample_pos_cs, sizeof(Eigen::Vector3d), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(iter_device, &i, sizeof(int), hipMemcpyHostToDevice));
    
    
    trace_backwards<<<blocks, cudathreads>>>(
          image, image_unredist, image_redist,
          redist_weight_per_pixel, unredist_weight_per_pixel, zbuffer,
          *sample_pos_cs_device, *focal_length_device, *aperture_radius_device, 
          *focus_distance_device, *sensor_width_device, *frame_aspect_ratio_device,
          *xres_device, *yres_device,
          *sample_device, *inv_density_device,
          *iter_device
    );

    // cleanup
    checkCudaErrors(hipFree(inv_density_device));
    checkCudaErrors(hipFree(sample_device));
    checkCudaErrors(hipFree(sample_pos_cs_device));
    checkCudaErrors(hipFree(iter_device));
  }

  checkCudaErrors(hipDeviceSynchronize());

  // cleanup
  checkCudaErrors(hipFree(xres_device));
  checkCudaErrors(hipFree(yres_device));
  checkCudaErrors(hipFree(focal_length_device));
  checkCudaErrors(hipFree(aperture_radius_device));
  checkCudaErrors(hipFree(focus_distance_device));
  checkCudaErrors(hipFree(sensor_width_device));
  checkCudaErrors(hipFree(frame_aspect_ratio_device));
  




// CLOSE


  std::vector<float> imageexr(yres * xres * 4);
  int offset = -1;

  for(unsigned px = 0; px < xres * yres; px++){ 

    Eigen::Vector4d redist = image_redist[px] / ((redist_weight_per_pixel[px] == 0.0) ? 1.0 : redist_weight_per_pixel[px]);
    Eigen::Vector4d unredist = image_unredist[px] / ((unredist_weight_per_pixel[px] == 0.0) ? 1.0 : unredist_weight_per_pixel[px]);
    Eigen::Vector4d combined_redist_unredist = (unredist * (1.0-redist_weight_per_pixel[px])) + (redist * (redist_weight_per_pixel[px]));

    if (combined_redist_unredist(3) > 0.95) combined_redist_unredist /= combined_redist_unredist(3);

    imageexr[++offset] = combined_redist_unredist(0);
    imageexr[++offset] = combined_redist_unredist(1);
    imageexr[++offset] = combined_redist_unredist(2);
    imageexr[++offset] = combined_redist_unredist(3);
  
  }

  // replace <aov> and <frame>
  std::string path = bidir_output_path;
  std::string path_replaced_aov = replace_first_occurence(path, "<aov>", "RGBA");
  
  std::string frame_str = std::to_string(framenumber);
  std::string frame_padded = std::string(4 - frame_str.length(), '0') + frame_str;
  std::string path_replaced_framenumber = replace_first_occurence(path, "<frame>", frame_padded);

  // dump framebuffers to exrs
  save_to_exr_rgba(imageexr, path_replaced_framenumber, xres, yres);

  printf("written image!");


  // cleanup
  hipFree(image);
  hipFree(image_unredist);
  hipFree(image_redist);
  hipFree(redist_weight_per_pixel);
  hipFree(unredist_weight_per_pixel);
  hipFree(zbuffer);
}